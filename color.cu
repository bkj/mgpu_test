#pragma GCC diagnostic ignored "-Wunused-result"
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/iterator/counting_iterator.h>
#include "thrust/random.h"

#define MANAGED

int n_rows;
int n_cols;
int n_nnz;

int* h_indptr;
int* h_indices;
float* h_data;

int* g_indptr;
int* g_indices;
float* g_data;

struct gpu_info {
  hipStream_t stream;
  hipEvent_t  event;
};

std::vector<gpu_info> infos;

hipStream_t master_stream;


struct my_timer_t {
  float time;

  my_timer_t() {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_);
  }

  ~my_timer_t() {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  // Alias of each other, start the timer.
  void begin() { hipEventRecord(start_); }
  void start() { this->begin(); }

  float end() {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&time, start_, stop_);

    return milliseconds();
  }

  float seconds() { return time * 1e-3; }
  float milliseconds() { return time; }

 private:
  hipEvent_t start_, stop_;
};

template <typename index_t, typename iterator_t>
void uniform_distribution(index_t begin, index_t end, iterator_t input) {
  using type_t = typename std::iterator_traits<iterator_t>::value_type;

  auto generate_random = [] __device__(int i) -> type_t {
    thrust::default_random_engine rng;
    rng.discard(i);
    return rng();
  };
  
  thrust::transform(thrust::make_counting_iterator(begin), thrust::make_counting_iterator(end), input, generate_random);
}

int get_num_gpus() {
  int num_gpus = -1;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

void enable_peer_access() {
  int num_gpus = get_num_gpus();
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for(int j = 0; j < num_gpus; j++) {
      if(i == j) 
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }
  
  hipSetDevice(0);
}

void create_contexts() {
  int num_gpus = get_num_gpus();
  
  hipSetDevice(0);
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreate(&info.event);
    infos.push_back(info);
  }
  
  hipSetDevice(0);
}

void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&n_rows, sizeof(int), 1, file);
  err = fread(&n_cols, sizeof(int), 1, file);
  err = fread(&n_nnz,  sizeof(int), 1, file);

  std::cout << "n_rows: " << n_rows << std::endl;
  std::cout << "n_cols: " << n_cols << std::endl;
  std::cout << "n_nnz: " << n_nnz << std::endl;

  hipMallocManaged(&g_indptr,  (n_rows + 1) * sizeof(int));
  hipMallocManaged(&g_indices, n_nnz        * sizeof(int));
  hipMallocManaged(&g_data,    n_nnz        * sizeof(float));

  err = fread(g_indptr,  sizeof(int),   n_rows + 1, file);
  err = fread(g_indices, sizeof(int),   n_nnz,      file);
  err = fread(g_data,    sizeof(float), n_nnz,      file);

#ifdef MANAGED
  hipMemAdvise(g_indptr,  (n_rows + 1) * sizeof(int),   hipMemAdviseSetReadMostly, 0);
  hipMemAdvise(g_indices, n_nnz        * sizeof(int),   hipMemAdviseSetReadMostly, 0);
  hipMemAdvise(g_data,    n_nnz        * sizeof(float), hipMemAdviseSetReadMostly, 0);
#endif  
}

void do_test() {
  srand(123123123);
  
  int num_gpus = get_num_gpus();

  // --
  // initialize frontier
  
  thrust::host_vector<int> h_input(n_rows);
  thrust::host_vector<int> h_output(n_rows);
  for(int i = 0; i < n_rows; i++) h_input[i] = i;
  for(int i = 0; i < n_rows; i++) h_output[i] = -1;

  thrust::device_vector<int> input   = h_input;
  thrust::device_vector<int> output  = h_output;
  
  // --
  // initialize data structures
  
  thrust::device_vector<int> d_colors;
  d_colors.resize(n_rows);
  thrust::fill(thrust::device, d_colors.begin(), d_colors.end(), -1);

  int* h_randoms = (int*)malloc(n_rows * sizeof(int));
  for(int i = 0; i < n_rows; i++) h_randoms[i] = rand();
  
  int* randoms;
  hipMallocManaged(&randoms, n_rows * sizeof(int));
  hipMemcpy(randoms, h_randoms, n_rows * sizeof(int), hipMemcpyHostToDevice);
#ifdef MANAGED
  hipMemAdvise(randoms, n_rows * sizeof(int), hipMemAdviseSetReadMostly, 0);
#endif

  int* colors  = d_colors.data().get();
  
  // --
  // Run
  
  hipSetDevice(0);  
  hipDeviceSynchronize();
  my_timer_t t;
  t.begin();

  int new_sizes[num_gpus];
  
  int* indptr  = g_indptr;
  int* indices = g_indices;
  float* data  = g_data;
  
  nvtxRangePushA("thrust_work");
  
  int iteration = 0;
  while(input.size() > 4) {
    
    int chunk_size  = (input.size() + num_gpus - 1) / num_gpus;
    
    #pragma omp parallel for num_threads(num_gpus)
    for(int i = 0 ; i < num_gpus ; i++) {
      
      hipSetDevice(i);

      auto fn = [indptr, indices, data, colors, randoms, iteration] __host__ __device__(int const& vertex) -> bool {
        int start  = indptr[vertex];
        int end    = indptr[vertex + 1];
        int degree = end - start;

        bool colormax = true;
        bool colormin = true;
        int color     = iteration * 2;

        for (int i = 0; i < degree; i++) {
          int u = indices[start + i];

          if (colors[u] != -1 && (colors[u] != color + 1) && (colors[u] != color + 2) || (vertex == u))
            continue;
          if (randoms[vertex] <= randoms[u])
            colormax = false;
          if (randoms[vertex] >= randoms[u])
            colormin = false;
        }

        if (colormax) {
          colors[vertex] = color + 1;
          return false;
        } else if (colormin) {
          colors[vertex] = color + 2;
          return false;
        } else {
          return true;
        }
      };
      
      auto input_begin  = input.begin() + chunk_size * i;
      auto input_end    = input.begin() + chunk_size * (i + 1);
      auto output_begin = output.begin() + chunk_size * i;
      if(i == num_gpus - 1) input_end = input.end();
      
      auto new_output_end = thrust::copy_if(
        thrust::cuda::par.on(infos[i].stream),
        input_begin,
        input_end,
        output_begin,
        fn
      );
      new_sizes[i] = (int)thrust::distance(output_begin, new_output_end);
      hipEventRecord(infos[i].event, infos[i].stream);
    }
    
    for(int i = 0; i < num_gpus; i++)
      hipStreamWaitEvent(master_stream, infos[i].event, 0);
    hipStreamSynchronize(master_stream);
    
    int total_length = 0;
    int offsets[num_gpus];
    offsets[0] = 0;
    for(int i = 1 ; i < num_gpus ; i++) offsets[i] = new_sizes[i - 1] + offsets[i - 1];
    for(int i = 0 ; i < num_gpus ; i++) total_length += new_sizes[i];

    // Reduce
    hipSetDevice(0);
    for(int i = 0; i < num_gpus; i++) {
      auto output_begin = output.begin() + chunk_size * i;
      thrust::copy_n(
        thrust::cuda::par.on(infos[0].stream),
        output_begin, 
        new_sizes[i], 
        input.begin() + offsets[i]
      );
    }
    
    hipEventRecord(infos[0].event, infos[0].stream);
    hipStreamWaitEvent(master_stream, infos[0].event, 0);
    hipStreamSynchronize(master_stream);
    
    input.resize(total_length);
    output.resize(total_length);
      
    iteration++;
    // t.end(); std::cout << "elapsed: " << t.milliseconds() << std::endl;
  }
  nvtxRangePop();
  
  // Log
  thrust::host_vector<int> out = d_colors;
  thrust::copy(out.begin(), out.begin() + 32, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
  
  hipSetDevice(0);
  t.end();  
  std::cout << "total_elapsed: " << t.milliseconds() << std::endl;
}

int main(int argc, char** argv) {
  std::string inpath = argv[1];
  
  enable_peer_access();
  create_contexts();
  read_binary(inpath);

  int num_gpus = get_num_gpus();
  std::cout << "color | num_gpus: " << num_gpus << std::endl;

  int num_iters = 4;
  for(int i = 0; i < num_iters; i++)
    do_test();
  
  std::cout << "-----" << std::endl;
  return EXIT_SUCCESS;
}