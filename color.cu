#pragma GCC diagnostic ignored "-Wunused-result"
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/iterator/counting_iterator.h>
#include "thrust/random.h"

int n_rows;
int n_cols;
int n_nnz;

int* h_indptr;
int* h_indices;

int* g_indptr;
int* g_indices;

int** all_indptrs;
int** all_indices;
int** all_randoms;
int** all_inputs;
int** all_colors;

struct gpu_info {
  hipStream_t stream;
  hipEvent_t  event;
};

std::vector<gpu_info> infos;

hipStream_t master_stream;


struct my_timer_t {
  float time;

  my_timer_t() {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_);
  }

  ~my_timer_t() {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  // Alias of each other, start the timer.
  void begin() { hipEventRecord(start_); }
  void start() { this->begin(); }

  float end() {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&time, start_, stop_);

    return milliseconds();
  }

  float seconds() { return time * 1e-3; }
  float milliseconds() { return time; }

 private:
  hipEvent_t start_, stop_;
};

int get_num_gpus() {
  int num_gpus = -1;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

void enable_peer_access() {
  int num_gpus = get_num_gpus();
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for(int j = 0; j < num_gpus; j++) {
      if(i == j) 
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }
  
  hipSetDevice(0);
}

void create_contexts() {
  int num_gpus = get_num_gpus();
  
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreateWithFlags(&info.event, hipEventDisableTiming);
    infos.push_back(info);
  }
  
  hipSetDevice(0);
}

void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&n_rows, sizeof(int), 1, file);
  err = fread(&n_cols, sizeof(int), 1, file);
  err = fread(&n_nnz,  sizeof(int), 1, file);

  h_indptr  = (int*  )malloc((n_rows + 1) * sizeof(int));
  h_indices = (int*  )malloc(n_nnz        * sizeof(int));

  err = fread(h_indptr,  sizeof(int),   n_rows + 1, file);
  err = fread(h_indices, sizeof(int),   n_nnz,      file);
  
  int num_gpus = get_num_gpus();
  
  all_indptrs = (int**)malloc(num_gpus * sizeof(int*));
  all_indices = (int**)malloc(num_gpus * sizeof(int*));
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    
    int* l_indptrs;
    int* l_indices;
    hipMalloc(&l_indptrs,  (n_rows + 1) * sizeof(int));
    hipMalloc(&l_indices,  (n_nnz     ) * sizeof(int));
    
    hipMemcpy(l_indptrs, h_indptr, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(l_indices, h_indices, n_nnz * sizeof(int), hipMemcpyHostToDevice);
    
    all_indptrs[i] = l_indptrs;
    all_indices[i] = l_indices;
  }
  hipSetDevice(0);

}

void do_test() {
  srand(345345345);
  
  int num_gpus = get_num_gpus();

  // --
  // initialize frontier
  
  int chunk_size  = (n_rows + num_gpus - 1) / num_gpus;
  
  
  // Inputs, chunked across devices
  all_inputs = (int**)malloc(num_gpus * sizeof(int*));
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    
    int* h_input = (int*)malloc(chunk_size * sizeof(int));
    
    int begin    = chunk_size * i;
    for(int ii = 0; ii < chunk_size ; ii++) {
      if(begin + ii < n_rows)
        h_input[ii] = begin + ii;
      else
        h_input[ii] = -1;
    }
    
    int* l_inputs;
    hipMalloc(&l_inputs, chunk_size * sizeof(int));
    hipMemcpy(l_inputs, h_input, chunk_size * sizeof(int), hipMemcpyHostToDevice);  
    all_inputs[i] = l_inputs;
  }
  hipSetDevice(0);
  
  
  // randoms, copied across devices
  int* h_randoms = (int*)malloc(n_rows * sizeof(int));
  for(int i = 0; i < n_rows; i++) h_randoms[i] = (int)rand();
  
  all_randoms = (int**)malloc(num_gpus * sizeof(int*));
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    
    int* l_randoms;
    hipMalloc(&l_randoms, n_rows * sizeof(int));
    hipMemcpy(l_randoms, h_randoms, n_rows * sizeof(int), hipMemcpyHostToDevice);  
    all_randoms[i] = l_randoms;
  }
  hipSetDevice(0);


  // colors, chunked across devices  
  all_colors = (int**)malloc(num_gpus * sizeof(int*));

  int* h_color = (int*)malloc(chunk_size * sizeof(int));
  for(int ii = 0; ii < chunk_size ; ii++) h_color[ii] = -1;
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    
    int* l_colors;
    hipMalloc(&l_colors, chunk_size * sizeof(int));
    hipMemcpy(l_colors, h_color, chunk_size * sizeof(int), hipMemcpyHostToDevice);  
    all_colors[i] = l_colors;
  }
  hipSetDevice(0);
  
  // --
  // Run
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);  
    hipDeviceSynchronize();
  }

  hipSetDevice(0);

  my_timer_t t;
  std::vector<float> per_iteration_times;

  int* colors0 = all_colors[0];
  int* colors1 = all_colors[1];
  int* colors2 = all_colors[2];
  int* colors3 = all_colors[3];
  
  nvtxRangePushA("thrust_work");

  int iteration = 0;
  while(iteration < 16) {
    t.begin();
    
    #pragma omp parallel for 
    for(int i = num_gpus - 1 ; i >= 0; i--) {
      
      hipSetDevice(i);

      int* indptr  = all_indptrs[i];
      int* indices = all_indices[i];
      int* randoms = all_randoms[i];
      int* inputs  = all_inputs[i];
      
      int offset   = i * chunk_size;
      int* wcolors = all_colors[i];
      
      auto fn = [indptr, indices, randoms, wcolors, offset, iteration, colors0, colors1, colors2, colors3, chunk_size] __host__ __device__(int const& vertex) {
        if(vertex == -1) return -1;
        
        int start  = indptr[vertex];
        int end    = indptr[vertex + 1];
        int degree = end - start;

        bool colormax = true;
        bool colormin = true;
        int color     = iteration * 2;

        int rv = randoms[vertex];
        
        for (int i = 0; i < degree; i++) {
          int u = indices[start + i];

          int ncolor = -1;
          if(u < chunk_size) {
            ncolor = colors0[u - 0 * chunk_size];
          } else if(u < 2 * chunk_size) {
            ncolor = colors1[u - 1 * chunk_size];
          } else if(u < 3 * chunk_size) {
            ncolor = colors2[u - 2 * chunk_size];
          } else {
            ncolor = colors3[u - 3 * chunk_size];
          }
          
          if (ncolor != -1 && (ncolor != color + 1) && (ncolor != color + 2) || (vertex == u)) continue;
          
          int ru = randoms[u];
          if(colormax) {if (rv <= ru) colormax = false;}
          if(colormin) {if (rv >= ru) colormin = false;}
          
          // if(!colormax && !colormin) break; // optimization
        }

        if (colormax) {
          wcolors[vertex - offset] = color + 1;
          return -1;
        } else if (colormin) {
          wcolors[vertex - offset] = color + 2;
          return -1;
        } else {
          return vertex;
        }
      };

      thrust::transform(
        thrust::cuda::par.on(infos[i].stream),
        inputs,
        inputs + chunk_size,
        inputs,
        fn
      );

      hipEventRecord(infos[i].event, infos[i].stream);
    }
    
    hipSetDevice(0);
    for(int i = 0; i < num_gpus; i++)
      hipStreamWaitEvent(master_stream, infos[i].event, 0);

    hipStreamSynchronize(master_stream);
      
    iteration++;
    t.end();
    per_iteration_times.push_back(t.milliseconds());
    std::cout << t.milliseconds() << std::endl;
  }
  nvtxRangePop();
  
  hipSetDevice(0);

  float total_elapsed = 0;
  for (auto& n : per_iteration_times)
    total_elapsed += n;

  std::cout << "total_elapsed: " << total_elapsed << std::endl;
}

int main(int argc, char** argv) {
  hipSetDevice(0);
  
  std::string inpath = argv[1];
  
  enable_peer_access();
  create_contexts();
  read_binary(inpath);

  int num_gpus = get_num_gpus();
  std::cout << "color | num_gpus: " << num_gpus << " vertices: " << n_rows << std::endl;

  int num_iters = 2;
  for(int i = 0; i < num_iters; i++)
    do_test();
  
  std::cout << "-----" << std::endl;
  return EXIT_SUCCESS;
}
