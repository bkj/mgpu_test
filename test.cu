#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"


void do_test(int num_arguments, char** argument_array) {
  srand(112233);
  
  // --
  // Create data
  
  int n = 1000000; // !!
  
  thrust::host_vector<int> h_input(n);
  thrust::host_vector<int> h_output(n);
  
  for(int i = 0; i < n; i++) 
    h_input[i] = rand() % 100000;
  
  thrust::fill(thrust::host, h_output.begin(), h_output.end(), -1);

  thrust::device_vector<int> input  = h_input;
  thrust::device_vector<int> output = h_output;

  // --
  // Setup data
  
  int num_gpus = 1;
  hipGetDeviceCount(&num_gpus);
  
  // Peer access
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for(int j = 0; j < num_gpus; j++) {
      if(i == j) 
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }
  
  // --
  // Setup devices
  
  hipSetDevice(0);
  hipStream_t master_stream;
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);

  auto chunk_size = (n + num_gpus - 1) / num_gpus;
  std::cout << "num_gpus  : " << num_gpus << std::endl;
  std::cout << "chunk_size: " << chunk_size << std::endl;
  
  struct gpu_info {
    hipStream_t stream;
    hipEvent_t  event;
  };
  
  std::vector<gpu_info> infos;
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreate(&info.event);
    infos.push_back(info);
  }
  
  // --
  // Thrust
  
  hipSetDevice(0);  
  
  auto fn = [=] __host__ __device__(int const& i) -> bool {
    int acc = 0;
    for(int ii = 0; ii < i; ii++)
      acc += ii;
    
    return (i + acc) % 2 == 0;
  };
  
  hipDeviceSynchronize();
  
  nvtxRangePushA("thrust_work");
  #pragma omp parallel for num_threads(num_gpus)
  for(int i = 0 ; i < num_gpus ; i++) {
    hipSetDevice(i);

    auto input_begin  = input.begin() + chunk_size * i;
    auto input_end    = input.begin() + chunk_size * (i + 1);
    auto output_begin = output.begin() + chunk_size * i;
    
    if(i == num_gpus - 1)
      input_end = input.end();
    
    thrust::copy_if(
      thrust::cuda::par.on(infos[i].stream),
      input_begin,
      input_end,
      output_begin,
      fn
    );
    hipEventRecord(infos[i].event, infos[i].stream);
  }
  
  for(int i = 0; i < num_gpus; i++)
    hipStreamWaitEvent(master_stream, infos[i].event, 0);
  
  for(int i = 0; i < num_gpus; i++) {hipSetDevice(i); hipDeviceSynchronize();}
  nvtxRangePop();

  thrust::host_vector<int> ttmp = output;
  thrust::copy(ttmp.begin(), ttmp.begin() + 100, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;


}

int main(int argc, char** argv) {
  for(int i = 0 ; i < 10 ; i++)
    do_test(argc, argv);
  return EXIT_SUCCESS;
}
