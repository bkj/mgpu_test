#include "hip/hip_runtime.h"
#include <cstdlib>  // EXIT_SUCCESS
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

__global__ void fn_kernel(int n, int* x, int* y) { 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    int i   = x[idx];
    int acc = 0;
    for(int ii = 0; ii < i; ii++) {
      acc += ii;
    }
    y[i] = (int)(acc % 2);
  }
}

void do_test(int num_arguments, char** argument_array) {
  
  // --
  // Create data
  
  int n = 200000;
  
  thrust::host_vector<int> h_input(n);
  thrust::host_vector<int> h_output(n);
  
  for(int i = 0; i < n; i++) h_input[i] = i;
  thrust::fill(thrust::host, h_output.begin(), h_output.end(), -1);
    
  // --
  // Setup data
  
  int num_gpus = 4;
  
  hipSetDevice(0);
  thrust::device_vector<int> input0  = h_input;
  thrust::device_vector<int> toutput0 = h_output;
  thrust::device_vector<int> koutput0 = h_output;

  hipSetDevice(1);
  thrust::device_vector<int> input1  = h_input;
  thrust::device_vector<int> toutput1 = h_output;
  thrust::device_vector<int> koutput1 = h_output;

  hipSetDevice(2);
  thrust::device_vector<int> input2  = h_input;
  thrust::device_vector<int> toutput2 = h_output;
  thrust::device_vector<int> koutput2 = h_output;

  hipSetDevice(3);
  thrust::device_vector<int> input3  = h_input;
  thrust::device_vector<int> toutput3 = h_output;
  thrust::device_vector<int> koutput3 = h_output;

  std::vector<thrust::device_vector<int>*> all_inputs;
  all_inputs.push_back(&input0);
  all_inputs.push_back(&input1);
  all_inputs.push_back(&input2);
  all_inputs.push_back(&input3);
  
  std::vector<thrust::device_vector<int>*> all_outputs_thrust;
  all_outputs_thrust.push_back(&toutput0);
  all_outputs_thrust.push_back(&toutput1);
  all_outputs_thrust.push_back(&toutput2);
  all_outputs_thrust.push_back(&toutput3);

  std::vector<thrust::device_vector<int>*> all_outputs_kernel;
  all_outputs_kernel.push_back(&koutput0);
  all_outputs_kernel.push_back(&koutput1);
  all_outputs_kernel.push_back(&koutput2);
  all_outputs_kernel.push_back(&koutput3);

  // --
  // Setup devices
  
  hipSetDevice(0);
  hipStream_t master_stream;
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);

  auto chunk_size = n / num_gpus;
  std::cout << "num_gpus  : " << num_gpus << std::endl;
  std::cout << "chunk_size: " << chunk_size << std::endl;
  
  struct gpu_info {
    hipStream_t stream;
    hipEvent_t  event;
  };
  
  std::vector<gpu_info> infos;
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreate(&info.event);
    infos.push_back(info);
  }
  
  // --
  // Run
  
  auto fn = [=] __host__ __device__(int const& i) -> bool {
    int acc = 0;
    for(int ii = 0; ii < i; ii++)
      acc += ii;
    
    return acc % 2 == 0 ? 0 : 1;
  };
  
  // Thrust
  nvtxRangePushA("thrust_work");
  for(int i = 0 ; i < num_gpus ; i++) {
    hipSetDevice(i);

    thrust::transform(
      thrust::cuda::par.on(infos[i].stream),
      all_inputs[i]->begin(),
      all_inputs[i]->end(),
      all_outputs_thrust[i]->begin(),
      fn
    );
    hipEventRecord(infos[i].event, infos[i].stream);
  }
  
  for(int i = 0; i < num_gpus; i++) hipStreamWaitEvent(master_stream, infos[i].event, 0);
  
  nvtxRangePop();
  
  // Kernel
  nvtxRangePushA("kernel_work");
  for(int i = 0 ; i < num_gpus ; i++) {
    hipSetDevice(i);
    fn_kernel<<<(n + 255) / 256, 256, 0, infos[i].stream>>>(
      n, 
      all_inputs[i]->data().get(),
      all_outputs_kernel[i]->data().get()
    );

    hipEventRecord(infos[i].event, infos[i].stream);
  }
  
  for(int i = 0; i < num_gpus; i++) hipStreamWaitEvent(master_stream, infos[i].event, 0);
  
  nvtxRangePop();
  
  hipSetDevice(0);
  thrust::host_vector<int> ttmp = *all_outputs_thrust[0];
  thrust::copy(ttmp.begin(), ttmp.begin() + 100, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
  
  thrust::host_vector<int> ktmp = *all_outputs_kernel[0];
  thrust::copy(ktmp.begin(), ktmp.begin() + 100, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
}

int main(int argc, char** argv) {
  do_test(argc, argv);
  return EXIT_SUCCESS;
}
