#pragma GCC diagnostic ignored "-Wunused-result"
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include <algorithm>
#include <random>
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/iterator/counting_iterator.h>
#include "thrust/random.h"

#define MANAGED

struct my_timer_t {
  float time;

  my_timer_t() {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_);
  }

  ~my_timer_t() {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  // Alias of each other, start the timer.
  void begin() { hipEventRecord(start_); }
  void start() { this->begin(); }

  float end() {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&time, start_, stop_);

    return milliseconds();
  }

  float seconds() { return time * 1e-3; }
  float milliseconds() { return time; }

 private:
  hipEvent_t start_, stop_;
};

template <typename index_t, typename iterator_t>
void uniform_distribution(index_t begin, index_t end, iterator_t input) {
  using type_t = typename std::iterator_traits<iterator_t>::value_type;

  auto generate_random = [] __device__(int i) -> type_t {
    thrust::default_random_engine rng;
    rng.discard(i);
    return rng();
  };
  
  thrust::transform(thrust::make_counting_iterator(begin), thrust::make_counting_iterator(end), input, generate_random);
}

int n_rows;
int n_cols;
int n_nnz;

int* h_indptr;
int* h_indices;
float* h_data;

int* g_indptr;
int* g_indices;
float* g_data;

struct gpu_info {
  hipStream_t stream;
  hipEvent_t  event;
};

std::vector<gpu_info> infos;

hipStream_t master_stream;

int get_num_gpus() {
  int num_gpus = -1;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

void enable_peer_access() {
  int num_gpus = get_num_gpus();
  
  for(int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for(int j = 0; j < num_gpus; j++) {
      if(i == j) 
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }
  
  hipSetDevice(0);
}

void create_contexts() {
  int num_gpus = get_num_gpus();
  
  hipSetDevice(0);
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);
  
  for(int i = 0 ; i < num_gpus ; i++) {
    gpu_info info;
    hipSetDevice(i);
    hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
    hipEventCreate(&info.event);
    infos.push_back(info);
  }
  
  hipSetDevice(0);
}

void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&n_rows, sizeof(int), 1, file);
  err = fread(&n_cols, sizeof(int), 1, file);
  err = fread(&n_nnz,  sizeof(int), 1, file);

  h_indptr  = (int*  )malloc((n_rows + 1) * sizeof(int));
  h_indices = (int*  )malloc(n_nnz        * sizeof(int));
  h_data    = (float*)malloc(n_nnz        * sizeof(float));

  err = fread(h_indptr,  sizeof(int),   n_rows + 1, file);
  err = fread(h_indices, sizeof(int),   n_nnz,      file);
  err = fread(h_data,    sizeof(float), n_nnz,      file);

#ifdef MANAGED
  hipMallocManaged(&g_indptr,  (n_rows + 1) * sizeof(int));
  hipMallocManaged(&g_indices, n_nnz        * sizeof(int));
  hipMallocManaged(&g_data,    n_nnz        * sizeof(float));
#else
  hipMalloc(&g_indptr, (n_rows + 1) * sizeof(int));
  hipMalloc(&g_indices, n_nnz       * sizeof(int));
  hipMalloc(&g_data,    n_nnz       * sizeof(float));
#endif

  hipMemcpy(g_indptr, h_indptr, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g_indices, h_indices, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g_data, h_data, (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);

#ifdef MANAGED
  for(int i = 0; i < get_num_gpus(); i++) {
    hipMemAdvise(g_indptr, (n_rows + 1) * sizeof(int), hipMemAdviseSetReadMostly, i);
    hipMemAdvise(g_indices, n_nnz * sizeof(int), hipMemAdviseSetReadMostly, i);
    hipMemAdvise(g_data, n_nnz * sizeof(float), hipMemAdviseSetReadMostly, i);
    
    hipMemPrefetchAsync(g_indptr, (n_rows + 1) * sizeof(int), i);
    hipMemPrefetchAsync(g_indices, n_nnz * sizeof(int), i);
    hipMemPrefetchAsync(g_data, n_nnz * sizeof(float), i);
  }
#endif  
}

void do_test() {
  srand(123123123);
  
  int num_gpus = get_num_gpus();

  // --
  // initialize frontier
  
  std::vector<int> tmp;
  for(int i = 0; i < n_rows; i++) tmp.push_back(i);
  std::random_device rd;
  std::mt19937 g(rd());
  std::shuffle(tmp.begin(), tmp.end(), g);
  
  thrust::host_vector<int> h_input(n_rows);
  thrust::host_vector<int> h_output(n_rows);
  for(int i = 0; i < n_rows; i++) h_input[i] = tmp[i];
  for(int i = 0; i < n_rows; i++) h_output[i] = -1;

  thrust::device_vector<int> input   = h_input;
  thrust::device_vector<int> output  = h_output;
  
  // --
  // initialize data structures

  // int* h_colors = (int*)malloc(n_rows * sizeof(int));
  // for(int i = 0; i < n_rows; i++) h_colors[i] = -1;  
  // int* colors;
  // hipMallocManaged(&colors, n_rows * sizeof(int));
  // hipMemcpy(colors, h_colors, n_rows * sizeof(int), hipMemcpyHostToDevice);
  
  thrust::device_vector<int> d_colors;
  d_colors.resize(n_rows);
  thrust::fill(thrust::device, d_colors.begin(), d_colors.end(), -1);
  int* colors  = d_colors.data().get();

  int* h_randoms = (int*)malloc(n_rows * sizeof(int));
  for(int i = 0; i < n_rows; i++) h_randoms[i] = rand() % n_rows;
  
  int* randoms;
  hipMallocManaged(&randoms, n_rows * sizeof(int));
  hipMemcpy(randoms, h_randoms, n_rows * sizeof(int), hipMemcpyHostToDevice);
#ifdef MANAGED
  for(int i = 0; i < num_gpus; i++) {
    hipMemAdvise(randoms, n_rows * sizeof(int), hipMemAdviseSetReadMostly, i);
    hipMemPrefetchAsync(randoms, n_rows * sizeof(int), i);
  }
#endif
  
  // --
  // run
  
  hipSetDevice(0);  
  hipDeviceSynchronize();
  
  int new_sizes[num_gpus];
  
  int* indptr  = g_indptr;
  int* indices = g_indices;
  float* data  = g_data;
  
  nvtxRangePushA("thrust_work");
  
  int iteration = 0;
// while(input.size() > 4) {
while(iteration < 29) {
  // printf("iteration: %d\n", iteration);
  
  int chunk_size  = (input.size() + num_gpus - 1) / num_gpus;
  
  #pragma omp parallel for num_threads(num_gpus)
  for(int i = 0 ; i < num_gpus ; i++) {
    
    hipSetDevice(i);

    auto fn = [indptr, indices, data, colors, randoms, iteration] __host__ __device__(int const& vertex) {
      if(vertex == -1) return -1;
      
      int start  = indptr[vertex];
      int end    = indptr[vertex + 1];
      int degree = end - start;

      bool colormax = true;
      bool colormin = true;
      int color     = iteration * 2;

      for (int i = 0; i < degree; i++) {
        int u = indices[start + i];

        if (colors[u] != -1 && (colors[u] != color + 1) && (colors[u] != color + 2) || (vertex == u))
          continue;
        if (randoms[vertex] <= randoms[u])
          colormax = false;
        if (randoms[vertex] >= randoms[u])
          colormin = false;
      }

      if (colormax) {
        colors[vertex] = color + 1;
        return -1;
      } else if (colormin) {
        colors[vertex] = color + 2;
        return -1;
      } else {
        return vertex;
      }
    };
    
    auto input_begin  = input.begin() + chunk_size * i;
    auto input_end    = input.begin() + chunk_size * (i + 1);
    auto output_begin = output.begin() + chunk_size * i;
    if(i == num_gpus - 1) input_end = input.end();
    
    thrust::transform(
      thrust::cuda::par.on(infos[i].stream),
      input_begin,
      input_end,
      input_begin,
      fn
    );
    // new_sizes[i] = (int)thrust::distance(output_begin, new_output_end);
    hipEventRecord(infos[i].event, infos[i].stream);
  }
  
  for(int i = 0; i < num_gpus; i++)
    hipStreamWaitEvent(master_stream, infos[i].event, 0);
  hipStreamSynchronize(master_stream);
  
  // int total_length = 0;
  // int offsets[num_gpus];
  // offsets[0] = 0;
  // for(int i = 1 ; i < num_gpus ; i++) offsets[i] = new_sizes[i - 1] + offsets[i - 1];
  // for(int i = 0 ; i < num_gpus ; i++) total_length += new_sizes[i];

  // // Reduce
  // #pragma omp parallel for num_threads(num_gpus)
  // for(int i = 0; i < num_gpus; i++) {
  //   hipSetDevice(i);

  //   auto output_begin = output.begin() + chunk_size * i;
  //   thrust::copy_n(
  //     thrust::cuda::par.on(infos[i].stream),
  //     output_begin, 
  //     new_sizes[i], 
  //     input.begin() + offsets[i]
  //   );
    
  //   hipEventRecord(infos[i].event, infos[i].stream);
  // }
  
  // for(int i = 0; i < num_gpus; i++)
  //   hipStreamWaitEvent(master_stream, infos[i].event, 0);
  
  // hipStreamSynchronize(master_stream);
  
  // input.resize(total_length);
  // output.resize(total_length);
    
  iteration++;
}
  nvtxRangePop();
  
  // Log
  thrust::host_vector<int> out = d_colors;
  thrust::copy(out.begin(), out.begin() + 32, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
  
  hipSetDevice(0);
}

int main(int argc, char** argv) {
  std::string inpath = argv[1];
  
  enable_peer_access();
  create_contexts();
  read_binary(inpath);

  int num_gpus = get_num_gpus();

  my_timer_t t;
  t.begin();
  
  int num_iters = 10;
  for(int i = 0; i < num_iters; i++)
    do_test();
  
  t.end();
  
  std::cout << "elapsed: " << t.milliseconds() << std::endl;
  
  return EXIT_SUCCESS;
}